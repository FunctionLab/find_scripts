#include "hip/hip_runtime.h"
/*
 * mersenneTwisterCUDA.c
 *
 *  Created on: May 5, 2010
 *      Author: Federico Dal Castello
 */

#include <stdio.h>
#include <cuda/hip/hip_runtime.h>

#include "mersenneTwisterCUDA.h"

#include "mersenneTwister_kernel.cu"
#include "mtUtils.c"
#include "timer.c"


mt_struct MT[MT_RNG_COUNT];
//static uint32_t state[MT_NN];

double exec_time_cuda, exec_time_kernel, exec_time_copyin, exec_time_copyout, exec_time_fileio, exec_time_cudamalloc;

void initMTRef_cuda(const char *fname) {

	FILE* fd = 0;

	// open the file for binary read
	if ((fd = fopen(fname, "rb")) == 0)	{
		if(fd) {
			fclose (fd);
		}
//		shrCheckError(0, 1);
	}

    for (int i = 0; i < MT_RNG_COUNT; i++) {
        //Inline structure size for compatibility,
        //since pointer types are 8-byte on 64-bit systems (unused *state variable)
        if( !fread(MT + i, 16 /* sizeof(mt_struct) */ * sizeof(int), 1, fd) ) {
//	        shrCheckError(0, 1);
        }
    }

    fclose(fd);
}


// Load twister configurations
void loadMTGPU_cuda(const char *fname){
    FILE *fd = fopen(fname, "rb");
    if(!fd){
        printf("initMTGPU(): failed to open %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }
    if( !fread(h_MT, sizeof(h_MT), 1, fd) ){
        printf("initMTGPU(): failed to load %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }
    fclose(fd);
}

// Initialize/seed twister for current GPU context
void seedMTGPU_cuda(unsigned int seed){
    int i;
    // Need to be thread-safe
    mt_struct_stripped *MT = (mt_struct_stripped *)malloc(MT_RNG_COUNT * sizeof(mt_struct_stripped));

    for(i = 0; i < MT_RNG_COUNT; i++){
        MT[i]      = h_MT[i];
        MT[i].seed = seed;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(ds_MT), MT, sizeof(h_MT));

    free(MT);
}


void initSamples_MT_CUDA(float *samples)
{
    float *d_Rand;	// device memory

    int r;

    unsigned int seed;
    unsigned int nPerRng = N/MT_RNG_COUNT;	// # of recurrence steps
    unsigned int num_results = MT_RNG_COUNT * nPerRng;	// Output size

	/* memory allocation on device */
	timer(1);
	hipMalloc((void **)&d_Rand, N * sizeof(float));
	exec_time_cudamalloc = timer(1);

    /* create and populate MT parameters array with initial pre-computed states*/
    srandom((unsigned int) (time(NULL)));
    double num = (random() / (float) RAND_MAX) * 4194304;
    seed = (unsigned int) trunc(num);
	timer(1);
	initMTRef_cuda("data/MersenneTwister.raw");
	loadMTGPU_cuda("data/MersenneTwister.dat");
	exec_time_fileio = timer(1);
	timer(1);
	seedMTGPU_cuda(seed);
        r = hipDeviceSynchronize();
	exec_time_copyin = timer(1);

	/* GPU random numbers generation */
	timer(1);
	mersenneTwister_kernel_cuda<<<128, 32>>>(d_Rand, nPerRng);
        r = hipDeviceSynchronize();
	exec_time_kernel = timer(1);

	/* read back the results */
	timer(1);
	hipMemcpy(samples, d_Rand, num_results * sizeof(float), hipMemcpyDeviceToHost);
        r = hipDeviceSynchronize();
	exec_time_copyout = timer(1);

	/* show some results */
//	for(int j=num_results-20; j < num_results; j++) {
//	for(int j=0; j < 30; j++) {
//		printf("result #%d: %f\n", j, samples[j]);
//	}

}


int main()
{
	float *samples;		// array of random samples
	float *dummy;		// dummy array - cudamalloc it to init device

	if(checkInputData()) {
		exit(EXIT_FAILURE);
	}

	samples  = (float *) malloc(sizeof(float) * N);
        for (int i = 0; i < N; i++) samples[i] = 0.0;

	/* Dummy cudamalloc to initialize device */
	hipMalloc((void **)&dummy, 32 * sizeof(float));

	printf("Computing over %d samples\n", N);


	/* Mersenne Twister on GPU with CUDA */
	printf("\n --- Mersenne Twister on GPU with CUDA ---\n");
	timer(0);
	initSamples_MT_CUDA(samples);
	double exec_time_cuda = timer(0);
	printf("cudamalloc time  = %f s\n", exec_time_cudamalloc);
	printf("file i/o time  = %f s\n", exec_time_fileio);
	printf("copyin time  = %f s\n", exec_time_copyin);
	printf("kernel exec time  = %f s\n", exec_time_kernel);
	printf("copyout time  = %f s\n", exec_time_copyout);
	printf("total exec time  = %f s\n", exec_time_cuda);
	printf("throughput = %f [MB/s]\n", computeThroughput(exec_time_cuda));

//	printMaxMin(samples);


	/* free up memory */
	free(samples);


	return EXIT_SUCCESS;
}

